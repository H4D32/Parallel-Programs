#include "hip/hip_runtime.h"
//
// Created by Zhong Yebin on 2023/9/16.
// Email: yebinzhong@link.cuhk.edu.cn
//
// CUDA implementation of transforming a JPEG image from RGB to gray
//

#include <iostream>

#include <hip/hip_runtime.h> // CUDA Header

#include "utils.hpp"

// CUDA kernel functon：RGB to Gray
__global__ void rgbSmooth(const unsigned char* input, unsigned char* output,
                          int width, int height, int num_channels,
                          const float* filter)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + width + 1;
    if (idx < width * height)
    {
        int a00 = (idx - width - 1) * num_channels; // idx - width - 1 -> Top Left
        int a10 = (idx - 1) * num_channels;         // idx - 1
        int a20 = (idx + width - 1) * num_channels; // idx + width - 1
        int a01 = (idx - width) * num_channels;     // idx - width
        int a11 = idx * num_channels;             // idx
        int a21 = (idx + width) * num_channels;     // idx + width
        int a02 = (idx - width + 1) * num_channels; // idx - width + 1
        int a12 = (idx + 1) * num_channels;         // idx + 1
        int a22 = (idx + width + 1) * num_channels; // idx + width + 1

        float sum_r = 0.0f, sum_g = 0.0f, sum_b = 0.0f;

        sum_r = static_cast<float>(input[a00]) * filter[0] +
                static_cast<float>(input[a01]) * filter[1] +
                static_cast<float>(input[a02]) * filter[2] +
                static_cast<float>(input[a10]) * filter[3] +
                static_cast<float>(input[a11]) * filter[4] +
                static_cast<float>(input[a12]) * filter[5] +
                static_cast<float>(input[a20]) * filter[6] +
                static_cast<float>(input[a21]) * filter[7] +
                static_cast<float>(input[a22]) * filter[8];

        sum_g = static_cast<float>(input[a00 + 1]) * filter[0] +
               static_cast<float>(input[a01 + 1]) * filter[1] +
               static_cast<float>(input[a02 + 1]) * filter[2] +
               static_cast<float>(input[a10 + 1]) * filter[3] +
               static_cast<float>(input[a11 + 1]) * filter[4] +
               static_cast<float>(input[a12 + 1]) * filter[5] +
               static_cast<float>(input[a20 + 1]) * filter[6] +
               static_cast<float>(input[a21 + 1]) * filter[7] +
               static_cast<float>(input[a22 + 1]) * filter[8];

        sum_b = static_cast<float>(input[a00 + 2]) * filter[0] +
               static_cast<float>(input[a01 + 2]) * filter[1] +
               static_cast<float>(input[a02 + 2]) * filter[2] +
               static_cast<float>(input[a10 + 2]) * filter[3] +
               static_cast<float>(input[a11 + 2]) * filter[4] +
               static_cast<float>(input[a12 + 2]) * filter[5] +
               static_cast<float>(input[a20 + 2]) * filter[6] +
               static_cast<float>(input[a21 + 2]) * filter[7] +
               static_cast<float>(input[a22 + 2]) * filter[8];

        output[(idx)*num_channels] = static_cast<unsigned char>(sum_r);
        output[(idx)*num_channels + 1] = static_cast<unsigned char>(sum_g);
        output[(idx)*num_channels + 2] = static_cast<unsigned char>(sum_b);
    }
}

int main(int argc, char** argv)
{
    // Verify input argument format
    if (argc != 3)
    {
        std::cerr << "Invalid argument, should be: ./executable "
                     "/path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }
    // Read from input JPEG
    const char* input_filepath = argv[1];
    std::cout << "Input file from: " << input_filepath << "\n";
    auto input_jpeg = read_from_jpeg(input_filepath);
    // Allocate memory on host (CPU)
    auto filteredImage = new unsigned char[input_jpeg.width * input_jpeg.height * input_jpeg.num_channels]; 
    // Allocate memory on device (GPU)
    unsigned char* d_input;
    unsigned char* d_output;
    float* d_filter;
    hipMalloc((void**)&d_input, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels *
                                     sizeof(unsigned char));
    hipMalloc((void**)&d_output, input_jpeg.width * input_jpeg.height *
                                      input_jpeg.num_channels *
                                      sizeof(unsigned char));
    hipMalloc((void**)&d_filter, 9 * sizeof(float));
    float array1DFilter[9] = {1.0f / 9, 1.0f / 9, 1.0f / 9, 1.0f / 9, 1.0f / 9,
                         1.0f / 9, 1.0f / 9, 1.0f / 9, 1.0f / 9};
    // Copy input data from host to device
    hipMemcpy(d_input, input_jpeg.buffer,
               input_jpeg.width * input_jpeg.height * input_jpeg.num_channels *
                   sizeof(unsigned char),
               hipMemcpyHostToDevice);
    hipMemcpy(d_filter, array1DFilter, 9 * sizeof(float),
               hipMemcpyHostToDevice);
    // Computation: RGB to Gray
    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int blockSize = 512; // 256
    // int numBlocks =
    //     (input_jpeg.width * input_jpeg.height + blockSize - 1) / blockSize;
    int numBlocks = (input_jpeg.width * input_jpeg.height) / blockSize + 1;
    hipEventRecord(start, 0); // GPU start time
    rgbSmooth<<<numBlocks, blockSize>>>(d_input, d_output, input_jpeg.width,
                                        input_jpeg.height,
                                        input_jpeg.num_channels, d_filter);
    hipEventRecord(stop, 0); // GPU end time
    hipEventSynchronize(stop);
    // Print the result of the GPU computation
    hipEventElapsedTime(&gpuDuration, start, stop);
    // Copy output data from device to host
    hipMemcpy(filteredImage, d_output,
               input_jpeg.width * input_jpeg.height * input_jpeg.num_channels * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    // Write GrayImage to output JPEG
    const char* output_filepath = argv[2];
    std::cout << "Output file to: " << output_filepath << "\n";
    JPEGMeta output_jpeg{filteredImage, input_jpeg.width, input_jpeg.height,
                         input_jpeg.num_channels, input_jpeg.color_space};
    if (write_to_jpeg(output_jpeg, output_filepath))
    {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }
    // Release allocated memory on device and host
    hipFree(d_input);
    hipFree(d_output);
    delete[] input_jpeg.buffer;
    delete[] filteredImage;
    std::cout << "Transformation Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}